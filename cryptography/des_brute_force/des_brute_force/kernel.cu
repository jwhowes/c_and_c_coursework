#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdint.h>

#include <stdio.h>

#include "des_consts.h"
#include "bit_functions.h"

using namespace std;

__device__ void f(uint64_t right, uint64_t key, uint64_t * ret) {
	// We're just assuming all the permutations work
	permute(right, E, ret, 48);  // E goes from length 32 to length 48 (not sure if this call will work)
	*ret ^= key;
	// Apply S-boxes
	for (int i = 0; i < 6; i++) {

	}
	// Apply P permutation
	permute(*ret, P, ret, 32);
}

__device__ void des_encrypt(uint64_t block, uint64_t * keys, uint64_t * ret) {

}

__global__ void brute_force_kernel(uint64_t plaintext, uint64_t ciphertext, uint64_t * res_key, bool * done) {
	// Generate first 3 bytes of the thread's key
	uint64_t thread_key = (uint64_t)(blockIdx.x * blockDim.x + threadIdx.x) << 35;
	//const uint64_t bit_mask = 0x00FFFFF800000000;
	uint64_t keys[16];
	uint64_t PC1_permuted;
	uint64_t C;
	uint64_t D;
	uint64_t temp;
	int v;
	// First three bytes of the thread key are now fixed
	for (uint64_t i = 0; i < thread_encryptions; i++) {
		if (*done) {
			return;
		}
		// Encrypt plaintext with thread_key
		// First, obtain key schedule
		permute(thread_key, PC_1, &PC1_permuted, 56);
		split_56(PC1_permuted, &C, &D);
		for (int j = 1; j <= 16; j++) {
			if (j == 1 || j == 2 || j == 9 || j == 16) {
				v = 1;
			}else {
				v = 2;
			}
			permute((C << 32) | D, PC_2, &keys[j - 1], 48);
			cycle_left(&C, v, 28);
			cycle_left(&D, v, 28);
		}
		des_encrypt(plaintext, keys, &temp);
		if (temp == ciphertext) {
			*done = true;
			*res_key = ciphertext;
			return;
		}
		thread_key++;
	}
}

int main() {
	uint64_t plaintext = 0x0123456789abcdef;
	uint64_t ciphertext = 0xa80f2c74f235484e;
	uint64_t plain_IP;
	permute(plaintext, IP, &plain_IP, 64);
	uint64_t cipher_FP;
	permute(ciphertext, FP, &cipher_FP, 64);
	plaintext = plain_IP;
	ciphertext = cipher_FP;
	// Instantiate CUDA variables
	uint64_t * res_key;
	bool * done;
	hipMalloc(&res_key, sizeof(uint64_t));
	hipMalloc(&done, sizeof(bool));
	hipMemcpy(done, false, sizeof(bool), hipMemcpyHostToDevice);
	brute_force_kernel <<<2048, 1024>>> (plaintext, ciphertext, res_key, done);
}