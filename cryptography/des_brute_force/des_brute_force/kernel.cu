#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdint.h>

#include <stdio.h>

#include "des_consts.h"
#include "bit_functions.h"

using namespace std;

__device__ void f(uint64_t right, uint64_t key, uint64_t * ret) {
	*ret = 0;
	const uint64_t row_mask = 0x21;
	const uint64_t col_mask = 0x1e;
	const uint64_t last_6 = 0x3f;
	uint64_t temp;
	// We're just assuming all the permutations work
	permute(right, E, &temp, 48, 32);  // E goes from length 32 to length 48 (not sure if this call will work)
	temp ^= key;
	// Apply S-boxes
	for (int i = 0; i < 8; i++) {
		int row = 0;
		int col = 0;
		// Get the last 6 bits of temp
		uint64_t row_bits = temp & row_mask;
		uint64_t col_bits = (temp & col_mask) >> 1;
		// Find row and column
		if (row_bits % 2 == 1) {
			row++;
		}
		if ((row_bits >> 5) % 2 == 1) {
			row += 2;
		}
		if (col_bits % 2 == 1) {
			col++;
		}
		col_bits >>= 1;
		if (col_bits % 2 == 1) {
			col += 2;
		}
		col_bits >>= 1;
		if (col_bits % 2 == 1) {
			col += 4;
		}
		col_bits >>= 1;
		if (col_bits % 2 == 1) {
			col += 8;
		}
		// Write the value of S box to *ret (I think maybe we have to write at the start, not at the end
		*ret |= S[i][row * 16 + col];
		*ret <<= 4;
		temp >>= 6;
	}
	*ret >>= 4;
	// Apply P permutation
	permute(*ret, P, ret, 32, 32);
}

__device__ void des_encrypt(uint64_t block, uint64_t * keys, uint64_t * ret) {
	uint64_t left;
	uint64_t right;
	uint64_t next_left;
	uint64_t f_temp;
	split_64(block, &left, &right);
	for (int i = 0; i < 16; i++) {
		next_left = right;
		f(right, keys[i], &f_temp);
		right = left ^ f_temp;
		left = next_left;
	}
	*ret = (right << 32) | left;
}

__global__ void brute_force_kernel(uint64_t plaintext, uint64_t ciphertext, uint64_t block_key, uint64_t * res_key, bool * done) {
	// Generate first 3 bytes of the thread's key
	uint64_t thread_key = block_key + ((uint64_t)(blockIdx.x * blockDim.x + threadIdx.x) << 35);
	//const uint64_t bit_mask = 0x00FFFFF800000000;
	uint64_t keys[16];
	uint64_t PC1_permuted;
	uint64_t C;
	uint64_t D;
	uint64_t temp;
	int v;
	// First three bytes of the thread key are now fixed
	if (*done) {
		return;
	}
	// Encrypt plaintext with thread_key
	// First, obtain key schedule
	permute(thread_key, PC_1, &PC1_permuted, 56, 56);
	split_56(PC1_permuted, &C, &D);
	for (int j = 1; j <= 16; j++) {
		if (j == 1 || j == 2 || j == 9 || j == 16) {
			v = 1;
		}else {
			v = 2;
		}
		permute((C << 28) | D, PC_2, &keys[j - 1], 48, 56);
		cycle_left(&C, v, 28);
		cycle_left(&D, v, 28);
	}
	des_encrypt(plaintext, keys, &temp);

	if (temp == ciphertext) {
		*done = true;
		*res_key = ciphertext;
		return;
	}
}


int main() {
	uint64_t plaintext = 0x0123456789abcdef;
	uint64_t ciphertext = 0xa80f2c74f235484e;
	uint64_t plain_IP;
	permute(plaintext, IP, &plain_IP, 64, 64);
	uint64_t cipher_FP;
	permute(ciphertext, FP, &cipher_FP, 64, 64);
	plaintext = plain_IP;
	ciphertext = cipher_FP;
	// Instantiate CUDA variables
	uint64_t * res_key;
	bool * done;
	hipMalloc(&res_key, sizeof(uint64_t));
	hipMalloc(&done, sizeof(bool));
	hipMemcpy(done, false, sizeof(bool), hipMemcpyHostToDevice);
	for (uint64_t i = 0; i < (uint64_t)1 << 46; i++) {
		brute_force_kernel <<<1, 1024>>> (plaintext, ciphertext, i << 46, res_key, done);
	}
	printf("All blocks started\n");
	hipDeviceSynchronize();
	return 0;
}