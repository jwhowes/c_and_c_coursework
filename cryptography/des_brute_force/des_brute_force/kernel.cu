#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdint.h>

#include <stdio.h>

#include "des_consts.h"
#include "bit_functions.h"

using namespace std;

__global__ void brute_force_kernel(uint64_t plaintext, uint64_t ciphertext, uint64_t * res_key, bool * done) {
	// Generate first 3 bytes of the thread's key
	uint64_t thread_id = (uint64_t)(blockIdx.x * blockDim.x + threadIdx.x);
	uint64_t thread_key = 0;
	uint64_t bit_mask = 0x7f;
	uint64_t keys[16];
	uint64_t PC1_permuted;
	uint64_t C;
	uint64_t D;
	int v;
	for (int i = 0; i < 3; i++) {
		uint64_t masked = thread_id & bit_mask;
		thread_key |= masked | (parities[masked] << 7);
		thread_key <<= 8;
		thread_id >>= 7;
	}
	thread_key <<= 32;
	// First three bytes of the thread key are now fixed
	for (uint64_t i = 0; i < thread_encryptions; i++) {
		thread_key >>= 40;
		uint64_t k = i;
		for (int j = 0; j < 5; j++) {
			uint64_t masked = k & bit_mask;
			thread_key <<= 8;
			thread_key |= masked | (parities[masked] << 7);
			k >>= 7;
		}
		// Encrypt plaintext with thread_key
		// First, obtain key schedule
		permute(thread_key, PC_1, &PC1_permuted, 64);
		split_64(PC1_permuted, &C, &D);
		for (int j = 1; j <= 16; j++) {
			if (j == 1 || j == 2 || j == 9 || j == 16) {
				v = 1;
			}else {
				v = 2;
			}
			permute((C << 32) | D, PC_2, &keys[j - 1], 64);
			cycle_left(&C, v, 32);
			cycle_left(&D, v, 32);
		}
	}
}

__device__ void des_encrypt(uint64_t block, uint64_t key, uint64_t * ret) {

}

int main() {
	uint64_t plaintext = 0x0123456789abcdef;
	uint64_t ciphertext = 0xa80f2c74f235484e;
	uint64_t plain_IP;
	permute(plaintext, IP, &plain_IP, 64);
	uint64_t cipher_FP;
	permute(ciphertext, FP, &cipher_FP, 64);
	plaintext = plain_IP;
	ciphertext = cipher_FP;
	// Instantiate CUDA variables
	uint64_t * res_key;
	bool * done;
	hipMalloc(&res_key, sizeof(uint64_t));
	hipMalloc(&done, sizeof(bool));
	hipMemcpy(done, false, sizeof(bool), hipMemcpyHostToDevice);
	brute_force_kernel <<<2048, 1024>>> (plaintext, ciphertext, res_key, done);
}